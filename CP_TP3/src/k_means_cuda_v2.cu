#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <time.h>

using namespace std;

int N;
int K;

typedef struct ponto{
    float x;
    float y;
    int cluster;
} Ponto;

typedef struct centroide{
    float x;
    float y;
    float soma_x;
    float soma_y;
    int total_pontos;
} Centroide;




__global__
void kMeansKernel (Ponto *d_points, Centroide *d_centroids) { // código executado no GPU
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    int N = 10000000;
    int K = 4;

    if (id < N){
        Ponto point = d_points[id];
        //d_points[id].cluster = 1;
        
        float d = 10000.0f;
        int cluster = -1;

        for(int j = 0; j < 4; j++){
            
            float tmp = (point.x - d_centroids[j].x) * (point.x- d_centroids[j].x) + 
                        (point.y - d_centroids[j].y) * (point.y - d_centroids[j].y);
            if (tmp < d){
                d = tmp;
                cluster = j;
            }
        }

        int cl = point.cluster; // obtençao do cluster em que estava o ponto

        if(cl != cluster && cluster != -1){
            d_points[id].cluster = cluster;
        }
    }

    if(id<K){
        d_centroids[id].soma_x = 0;
        d_centroids[id].soma_y = 0;
        d_centroids[id].total_pontos = 0;
    }
    
    int cluster = d_points[id].cluster;
    atomicAdd(&d_centroids[cluster].soma_x,d_points[id].x);
    atomicAdd(&d_centroids[cluster].soma_y,d_points[id].y);
    atomicAdd(&d_centroids[cluster].total_pontos,1);
    
    
    
}

__global__
void newCentroidsKernel (Ponto* d_points, Centroide* d_centroids){

    int id = threadIdx.x;
    
    d_centroids[id].x = d_centroids[id].soma_x/d_centroids[id].total_pontos;
    d_centroids[id].y = d_centroids[id].soma_y/d_centroids[id].total_pontos;


}



void init_pontos(Ponto** p, Centroide** c){
     
    *p = (Ponto *)malloc(sizeof( Ponto ) * N);
    *c = (Centroide *)malloc(sizeof( struct centroide ) * K);
    
    srand(10); //inicialização dos pontos
    for (int i = 0; i < N; i++){
        (*p)[i].x = (float) rand() / RAND_MAX;
        (*p)[i].y = (float) rand() / RAND_MAX;
        (*p)[i].cluster = -1;
    }
    
    for(int i  = 0; i < K ; i++){ // atribuição dos primeiros valores do centroid dos clusters
        (*c)[i].x = (*p)[i].x;
        (*c)[i].y = (*p)[i].y;
        (*c)[i].soma_x = (*p)[i].x;
        (*c)[i].soma_y = (*p)[i].y;
        (*c)[i].total_pontos = 1;
    }

    
}


void new_centroids(Ponto* points, Centroide* centroides){// para cada centroide, vai ser calculado o novo valor das suas coordenadas, com base na média geométrica dos pontos que estão no cluster correspondente
    
    //inicializar os centroides a 0 novamente
    for (int i = 0; i < K; i++){
        centroides[i].soma_x = 0;
        centroides[i].soma_y = 0;
        centroides[i].total_pontos = 0;
    }
    
    //calcular a soma de pontos e nº de pontos para cada centroide
    for(int i = 0; i < N; i++){
        int cluster = points[i].cluster;
        centroides[cluster].soma_x += points[i].x;
        centroides[cluster].soma_y += points[i].y;
        centroides[cluster].total_pontos++;
    }
    
    //calcular os novos centroides
    for(int i = 0; i < K; i++){
        centroides[i].x = centroides[i].soma_x / centroides[i].total_pontos;
        centroides[i].y = centroides[i].soma_y / centroides[i].total_pontos;
    }
}


void printPoints(Ponto* points, Centroide* centroids, int n){
    for(int i = 0; i < n; i++){
        printf("Ponto %d :: (%f,%f) -> %d\n", i, points[i].x, points[i].y, points[i].cluster);
    }
    for(int i = 0; i < 4; i++){
        printf("Centroide %d :: (%f,%f) -> %d\n", i, centroids[i].soma_x, centroids[i].soma_y, centroids[i].total_pontos);
    }
}


int k_means(Ponto* points, Centroide* centroids){

    //int changed_some_point = 1;
    int n_iter = 0;


    // pointer to the device memory
    Ponto *d_points;
    Centroide *d_centroids;


    // allocate memory on the device
    hipMalloc ( &d_points, N*sizeof(Ponto));
	hipMalloc ( &d_centroids, K*sizeof(Centroide));


    //copy points to device
    hipMemcpy(d_points, points, N*sizeof(Ponto), hipMemcpyHostToDevice);
    //copy centroids to device
    hipMemcpy(d_centroids, centroids, K*sizeof(Centroide), hipMemcpyHostToDevice);


    while(/*changed_some_point &&*/ n_iter < 20){// enquanto há pontos a mudarem de cluster
        //changed_some_point = 0;

        // launch the kernel
	    //startKernelTime ();
	    kMeansKernel <<< 20000, 500 >>> (d_points, d_centroids); // atribuit os pontos a centroides

        newCentroidsKernel <<< 1, 4 >>> (d_points, d_centroids); //calcular os novos centroides

        //printPoints(points, centroids, 20);
        //new_centroids(points, centroids);
        n_iter++;
    }

    hipMemcpy(centroids, d_centroids, K*sizeof(Centroide), hipMemcpyDeviceToHost);

    // free the device memory
    hipFree(d_points);
    hipFree(d_centroids);


    return n_iter;
}




int main(int argc, char** argv){
    Ponto * pontos;
    Centroide * centroides;
    
    N = atoi(argv[1]);
    K = atoi(argv[2]);
    
    init_pontos(&pontos, &centroides);

    //printPoints(pontos, centroides, 20);

    clock_t start, end;
    double elapsed;

    start = clock();

    int n_iter = k_means(pontos,centroides);

    end = clock();
    elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Elapsed time: %lf seconds\n", elapsed);
    
    printf("N = %d, K = %d\n", N, K);
    for(int i = 0; i < K; i++)
       printf("Center: (%f,%f) : Size: %d\n", centroides[i].x, centroides[i].y, centroides[i].total_pontos); 
    printf("Iterations: %d\n", n_iter);
    
}
