#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <time.h>

using namespace std;

int N;
int K;

typedef struct ponto{
    float x;
    float y;
    int cluster;
} Ponto;

typedef struct centroide{
    float x;
    float y;
    float soma_x;
    float soma_y;
    int total_pontos;
} Centroide;




__global__
void kMeansKernel (Ponto *d_points, Centroide *d_centroids) { // código executado no GPU
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    int N = 10000000;
    int K = 4;

    if (id < N){
        Ponto point = d_points[id];
        //d_points[id].cluster = 1;
        
        float d = 10000.0f;
        int cluster = -1;

        for(int j = 0; j < 4; j++){
            
            float tmp = (point.x - d_centroids[j].x) * (point.x- d_centroids[j].x) + 
                        (point.y - d_centroids[j].y) * (point.y - d_centroids[j].y);
            if (tmp < d){
                d = tmp;
                cluster = j;
            }
        }

        int cl = point.cluster; // obtençao do cluster em que estava o ponto

        if(cl != cluster && cluster != -1){
            d_points[id].cluster = cluster;
        }

        
        
    }

    __syncthreads();

    if(id<K){
        d_centroids[id].soma_x = 0;
        d_centroids[id].soma_y = 0;
        d_centroids[id].total_pontos = 0;
    }
    

}

__global__
void newCentroidsKernel (Ponto* d_points, Centroide* d_centroids){

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int N = 10000000;

    if (id < N){
        //get idx of thread at the block level
        const int s_idx = threadIdx.x;

        //put the datapoints and corresponding cluster assignments in shared memory so that they can be summed by thread 0 later
        __shared__ Ponto s_points[512];
        s_points[s_idx] = d_points[id];

        __syncthreads();

        if(s_idx == 0){

            float b_cluster_sums_x[4];
            float b_cluster_sums_y[4];
            int b_cluster_sizes[4]; 

            for(int i = 0; i < 4; i++){
                b_cluster_sums_x[i] = 0.0f;
                b_cluster_sums_y[i] = 0.0f;
                b_cluster_sizes[i] = 0;
            }
            
            int cluster = -1;

            for(int j = 0; j < blockDim.x && (blockIdx.x * blockDim.x + j) < 10000000 ; j++){

                cluster = s_points[j].cluster;
                b_cluster_sums_x[cluster] += s_points[j].x;
                b_cluster_sums_y[cluster] += s_points[j].y;
                b_cluster_sizes[cluster] += 1;

            }

            for(int j = 0; j < 4; j++){
                atomicAdd(&d_centroids[j].soma_x, b_cluster_sums_x[j]);
                atomicAdd(&d_centroids[j].soma_y, b_cluster_sums_y[j]);
                atomicAdd(&d_centroids[j].total_pontos, b_cluster_sizes[j]);
            }
        }

    }

    __syncthreads();

    if(id < 4){
        d_centroids[id].x = d_centroids[id].soma_x/d_centroids[id].total_pontos;
        d_centroids[id].y = d_centroids[id].soma_y/d_centroids[id].total_pontos;
    }     

}



void init_pontos(Ponto** p, Centroide** c){
     
    *p = (Ponto *)malloc(sizeof( Ponto ) * N);
    *c = (Centroide *)malloc(sizeof( struct centroide ) * K);
    
    srand(10); //inicialização dos pontos
    for (int i = 0; i < N; i++){
        (*p)[i].x = (float) rand() / RAND_MAX;
        (*p)[i].y = (float) rand() / RAND_MAX;
        (*p)[i].cluster = -1;
    }
    
    for(int i  = 0; i < K ; i++){ // atribuição dos primeiros valores do centroid dos clusters
        (*c)[i].x = (*p)[i].x;
        (*c)[i].y = (*p)[i].y;
        (*c)[i].soma_x = (*p)[i].x;
        (*c)[i].soma_y = (*p)[i].y;
        (*c)[i].total_pontos = 1;
    }

    
}


void new_centroids(Ponto* points, Centroide* centroides){// para cada centroide, vai ser calculado o novo valor das suas coordenadas, com base na média geométrica dos pontos que estão no cluster correspondente
    
    //inicializar os centroides a 0 novamente
    for (int i = 0; i < K; i++){
        centroides[i].soma_x = 0;
        centroides[i].soma_y = 0;
        centroides[i].total_pontos = 0;
    }
    
    //calcular a soma de pontos e nº de pontos para cada centroide
    for(int i = 0; i < N; i++){
        int cluster = points[i].cluster;
        centroides[cluster].soma_x += points[i].x;
        centroides[cluster].soma_y += points[i].y;
        centroides[cluster].total_pontos++;
    }
    
    //calcular os novos centroides
    for(int i = 0; i < K; i++){
        centroides[i].x = centroides[i].soma_x / centroides[i].total_pontos;
        centroides[i].y = centroides[i].soma_y / centroides[i].total_pontos;
    }
}


void printPoints(Ponto* points, Centroide* centroids, int n){
    for(int i = 0; i < n; i++){
        printf("Ponto %d :: (%f,%f) -> %d\n", i, points[i].x, points[i].y, points[i].cluster);
    }
    for(int i = 0; i < 4; i++){
        printf("Centroide %d :: (%f,%f) -> %d\n", i, centroids[i].soma_x, centroids[i].soma_y, centroids[i].total_pontos);
    }
}


int k_means(Ponto* points, Centroide* centroids){

    //int changed_some_point = 1;
    int n_iter = 0;


    // pointer to the device memory
    Ponto *d_points;
    Centroide *d_centroids;


    // allocate memory on the device
    hipMalloc ( &d_points, N*sizeof(Ponto));
	hipMalloc ( &d_centroids, K*sizeof(Centroide));


    //copy points to device
    hipMemcpy(d_points, points, N*sizeof(Ponto), hipMemcpyHostToDevice);
    //copy centroids to device
    hipMemcpy(d_centroids, centroids, K*sizeof(Centroide), hipMemcpyHostToDevice);


    while(/*changed_some_point &&*/ n_iter < 20){// enquanto há pontos a mudarem de cluster
        //changed_some_point = 0;

        // launch the kernel
	    //startKernelTime ();
	    kMeansKernel <<< 20000, 500 >>> (d_points, d_centroids); // atribuit os pontos a centroides

        newCentroidsKernel <<< (N+512-1)/512, 512 >>> (d_points, d_centroids); //calcular os novos centroides
        hipMemcpy(centroids, d_centroids, K*sizeof(Centroide), hipMemcpyDeviceToHost);
        for(int i = 0; i < K; i++)
            printf("Center: (%f,%f) : Size: %d\n", centroids[i].x, centroids[i].y, centroids[i].total_pontos);      

        //printPoints(points, centroids, 20);
        //new_centroids(points, centroids);
        n_iter++;
    }

    hipMemcpy(centroids, d_centroids, K*sizeof(Centroide), hipMemcpyDeviceToHost);

    // free the device memory
    hipFree(d_points);
    hipFree(d_centroids);


    return n_iter;
}




int main(int argc, char** argv){
    Ponto * pontos;
    Centroide * centroides;
    
    N = atoi(argv[1]);
    K = atoi(argv[2]);
    
    

    //printPoints(pontos, centroides, 20);

    clock_t start, end;
    double elapsed;

    start = clock();
    init_pontos(&pontos, &centroides);
    int n_iter = k_means(pontos,centroides);

    end = clock();
    elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Elapsed time: %lf seconds\n", elapsed);
    
    printf("N = %d, K = %d\n", N, K);
    for(int i = 0; i < K; i++)
       printf("Center: (%f,%f) : Size: %d\n", centroides[i].x, centroides[i].y, centroides[i].total_pontos); 
    printf("Iterations: %d\n", n_iter);
    
}
