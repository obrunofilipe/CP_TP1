#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <chrono>


using namespace std;

int N;
int K;

typedef struct ponto{
    float x;
    float y;
    int cluster;
} Ponto;

typedef struct centroide{
    float x;
    float y;
    float soma_x;
    float soma_y;
    int total_pontos;
} Centroide;




__global__
void kMeansKernel (Ponto *d_points, Centroide *d_centroids) { // código executado no GPU
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    Ponto point = d_points[id];
    //d_points[id].cluster = 1;
    
    float d = 10000.0f;
    int cluster = -1;

    for(int j = 0; j < 4; j++){
        
        float tmp = (point.x - d_centroids[j].x) * (point.x- d_centroids[j].x) + 
                    (point.y - d_centroids[j].y) * (point.y - d_centroids[j].y);
        if (tmp < d){
            d = tmp;
            cluster = j;
        }
    }

    int cl = point.cluster; // obtençao do cluster em que estava o ponto

    if(cl != cluster && cluster != -1){
        d_points[id].cluster = cluster;
    }
}

void launchKernel(Ponto *points, Centroide *centroids){

    // pointer to the device memory
    Ponto *d_points;
    Centroide *d_centroids;


    // allocate memory on the device
    hipMalloc ( &d_points, N*sizeof(Ponto));
	hipMalloc ( &d_centroids, K*sizeof(Centroide));
    //checkCUDAError("mem allocation");

    //copy points to device
    hipMemcpy(d_points, points, N*sizeof(Ponto), hipMemcpyHostToDevice);
    //copy centroids to device
    hipMemcpy(d_centroids, centroids, K*sizeof(Centroide), hipMemcpyHostToDevice);
    //checkCUDAError("memcpy h->d");


    // launch the kernel
	//startKernelTime ();
	kMeansKernel <<< 20000, 500 >>> (d_points, d_centroids);
	//stopKernelTime ();
	//checkCUDAError("kernel invocation");
    //cudaDeviceSynchronize();
    // copy points output back to host
    hipMemcpy(points, d_points, N*sizeof(Ponto), hipMemcpyDeviceToHost);
    // copy centroids output back to host
    hipMemcpy(centroids, d_centroids, K*sizeof(Centroide), hipMemcpyDeviceToHost);
    //checkCUDAError("memcpy d->h");


    // free the device memory
    hipFree(d_points);
    hipFree(d_centroids);
    //checkCUDAError("mem free");

    


}

void init_pontos(Ponto** p, Centroide** c){
     
    *p = (Ponto *)malloc(sizeof( Ponto ) * N);
    *c = (Centroide *)malloc(sizeof( struct centroide ) * K);
    
    srand(10); //inicialização dos pontos
    for (int i = 0; i < N; i++){
        (*p)[i].x = (float) rand() / RAND_MAX;
        (*p)[i].y = (float) rand() / RAND_MAX;
        (*p)[i].cluster = -1;
    }
    
    for(int i  = 0; i < K ; i++){ // atribuição dos primeiros valores do centroid dos clusters
        (*c)[i].x = (*p)[i].x;
        (*c)[i].y = (*p)[i].y;
        (*c)[i].soma_x = (*p)[i].x;
        (*c)[i].soma_y = (*p)[i].y;
        (*c)[i].total_pontos = 1;
    }

    
}


void new_centroids(Ponto* points, Centroide* centroides){// para cada centroide, vai ser calculado o novo valor das suas coordenadas, com base na média geométrica dos pontos que estão no cluster correspondente
    
    //inicializar os centroides a 0 novamente
    for (int i = 0; i < K; i++){
        centroides[i].soma_x = 0;
        centroides[i].soma_y = 0;
        centroides[i].total_pontos = 0;
    }
    
    //calcular a soma de pontos e nº de pontos para cada centroide
    for(int i = 0; i < N; i++){
        int cluster = points[i].cluster;
        centroides[cluster].soma_x += points[i].x;
        centroides[cluster].soma_y += points[i].y;
        centroides[cluster].total_pontos++;
    }
    
    //calcular os novos centroides
    for(int i = 0; i < K; i++){
        centroides[i].x = centroides[i].soma_x / centroides[i].total_pontos;
        centroides[i].y = centroides[i].soma_y / centroides[i].total_pontos;
    }
}


void printPoints(Ponto* points, Centroide* centroids, int n){
    for(int i = 0; i < n; i++){
        printf("Ponto %d :: (%f,%f) -> %d\n", i, points[i].x, points[i].y, points[i].cluster);
    }
    for(int i = 0; i < 4; i++){
        printf("Centroide %d :: (%f,%f) -> %d\n", i, centroids[i].soma_x, centroids[i].soma_y, centroids[i].total_pontos);
    }
}


int k_means(Ponto* points, Centroide* centroids){

    int changed_some_point = 1;
    int n_iter = 0;

    while(/*changed_some_point &&*/ n_iter < 20){// enquanto há pontos a mudarem de cluster
        //changed_some_point = 0;
        launchKernel(points, centroids);
        //printPoints(points, centroids, 20);
        new_centroids(points, centroids);
        n_iter++;
    }
    return n_iter;
}




int main(int argc, char** argv){
    Ponto * pontos;
    Centroide * centroides;
    
    N = atoi(argv[1]);
    K = atoi(argv[2]);
    
    init_pontos(&pontos, &centroides);

    //printPoints(pontos, centroides, 20);

    int n_iter = k_means(pontos,centroides);
    
    printf("N = %d, K = %d\n", N, K);
    for(int i = 0; i < K; i++)
       printf("Center: (%f,%f) : Size: %d\n", centroides[i].x, centroides[i].y, centroides[i].total_pontos); 
    printf("Iterations: %d\n", n_iter);
    
}
